
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main() {
    	print_kernel<<<100, 100>>>();
	hipError_t cudaerr = hipDeviceSynchronize();

}

